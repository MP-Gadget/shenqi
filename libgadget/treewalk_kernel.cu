#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>           // For CUDA runtime API functions.
#include <>  // To support device-related parameters.
// #include "treewalk.h"               // Include necessary header for TreeWalk structures and methods
#include "treewalk_kernel.h"
#include "gravshort.h"
// treewalk_kernel.cu
#include "shortrange-kernel_device.cu"
// #include "gravity.h"

#define NTAB_device (sizeof(shortrange_force_kernels) / sizeof(shortrange_force_kernels[0]))
/*! variables for short-range lookup table */
__device__ static float shortrange_table[NTAB_device], shortrange_table_potential[NTAB_device], shortrange_table_tidal[NTAB_device];

__device__ static double GravitySoftening_device = 0.0;

__device__ double FORCE_SOFTENING_device(void)
{
    // raise error if GravitySoftening_device is not set
    if (GravitySoftening_device == 0.0) {
        printf("GravitySoftening_device is not set!\n");
        return 0.0;
    }
    /* Force is Newtonian beyond this.*/
    return 2.8 * GravitySoftening_device;
}

/* multiply force factor (*fac) and potential (*pot) by the shortrange force window function*/
__device__ int
grav_apply_short_range_window_device(double r, double * fac, double * pot, const double cellsize)
{
    const double dx = shortrange_force_kernels[1][0];
    double i = (r / cellsize / dx);
    size_t tabindex = floor(i);
    if(tabindex >= NTAB_device - 1)
        return 1;
    /* use a linear interpolation; */
    *fac *= (tabindex + 1 - i) * shortrange_table[tabindex] + (i - tabindex) * shortrange_table[tabindex + 1];
    *pot *= (tabindex + 1 - i) * shortrange_table_potential[tabindex] + (i - tabindex) * shortrange_table_potential[tabindex];
    return 0;
}

/* Add the acceleration from a node or particle to the output structure,
 * computing the short-range kernel and softening.*/
__device__ static void
apply_accn_to_output_device(TreeWalkResultGravShort * output, const double dx[3], const double r2, const double mass, const double cellsize)
{
    const double r = sqrt(r2);

    const double h = FORCE_SOFTENING_device();
    double fac = mass / (r2 * r);
    double facpot = -mass / r;

    if(r2 < h*h)
    {
        double wp;
        const double h3_inv = 1.0 / h / h / h;
        const double u = r / h;
        if(u < 0.5) {
            fac = mass * h3_inv * (10.666666666667 + u * u * (32.0 * u - 38.4));
            wp = -2.8 + u * u * (5.333333333333 + u * u * (6.4 * u - 9.6));
        }
        else {
            fac =
                mass * h3_inv * (21.333333333333 - 48.0 * u +
                        38.4 * u * u - 10.666666666667 * u * u * u - 0.066666666667 / (u * u * u));
            wp =
                -3.2 + 0.066666666667 / u + u * u * (10.666666666667 +
                        u * (-16.0 + u * (9.6 - 2.133333333333 * u)));
        }
        facpot = mass / h * wp;
    }

    if(0 == grav_apply_short_range_window_device(r, &fac, &facpot, cellsize)) {
        int i;
        for(i = 0; i < 3; i++)
            output->Acc[i] += dx[i] * fac;
        output->Potential += facpot;
    }
}

__device__ static int
shall_we_discard_node_device(const double len, const double r2, const double center[3], const double inpos[3], const double BoxSize, const double rcut, const double rcut2)
{
    /* This checks the distance from the node center of mass
     * is greater than the cutoff. */
    if(r2 > rcut2)
    {
        /* check whether we can stop walking along this branch */
        const double eff_dist = rcut + 0.5 * len;
        int i;
        /*This checks whether we are also outside this region of the oct-tree*/
        /* As long as one dimension is outside, we are fine*/
        for(i=0; i < 3; i++)
            if(fabs(NEAREST(center[i] - inpos[i], BoxSize)) > eff_dist)
                return 1;
    }
    return 0;
}

__device__ static int
shall_we_open_node_device(const double len, const double mass, const double r2, const double center[3], const double inpos[3], const double BoxSize, const double aold, const int TreeUseBH, const double BHOpeningAngle2)
{
    /* Check the relative acceleration opening condition*/
    if((TreeUseBH == 0) && (mass * len * len > r2 * r2 * aold))
         return 1;

    double bhangle = len * len  / r2;
     /*Check Barnes-Hut opening angle*/
    if(bhangle > BHOpeningAngle2)
         return 1;

    const double inside = 0.6 * len;
    /* Open the cell if we are inside it, even if the opening criterion is not satisfied.*/
    if(fabs(NEAREST(center[0] - inpos[0], BoxSize)) < inside &&
        fabs(NEAREST(center[1] - inpos[1], BoxSize)) < inside &&
        fabs(NEAREST(center[2] - inpos[2], BoxSize)) < inside)
        return 1;

    /* ok, node can be used */
    return 0;
}

__device__ void
treewalk_add_counters_device(LocalTreeWalk * lv, const int64_t ninteractions)
{
    if(lv->maxNinteractions < ninteractions)
        lv->maxNinteractions = ninteractions;
    if(lv->minNinteractions > ninteractions)
        lv->minNinteractions = ninteractions;
    lv->Ninteractions += ninteractions;
}

__device__ int treewalk_export_particle_device(LocalTreeWalk * lv, int no)
{
    // if(lv->mode != TREEWALK_TOPTREE || no < lv->tw->tree->lastnode) {
    //     endrun(1, "Called export not from a toptree.\n");
    // }
    // if(!lv->DataIndexTable)
    //     endrun(1, "DataIndexTable not allocated, treewalk_export_particle called in the wrong way\n");
    // if(no - lv->tw->tree->lastnode > lv->tw->tree->NTopLeaves)
    //     endrun(1, "Bad export leaf: no = %d lastnode %d ntop %d target %d\n", no, lv->tw->tree->lastnode, lv->tw->tree->NTopLeaves, lv->target);
    const int target = lv->target;
    TreeWalk * tw = lv->tw;
    const int task = tw->tree->TopLeaves[no - tw->tree->lastnode].Task;
    /* This index is a unique entry in the global DataIndexTable.*/
    size_t nexp = lv->Nexport;
    /* If the last export was to this task, we can perhaps just add this export to the existing NodeList. We can
     * be sure that all exports of this particle are contiguous.*/
    if(lv->NThisParticleExport >= 1 && lv->DataIndexTable[nexp-1].Task == task) {
#ifdef DEBUG
        /* This is just to be safe: only happens if our indices are off.*/
        if(lv->DataIndexTable[nexp - 1].Index != target)
            endrun(1, "Previous of %ld exports is target %d not current %d\n", lv->NThisParticleExport, lv->DataIndexTable[nexp-1].Index, target);
#endif
        if(lv->nodelistindex < NODELISTLENGTH) {
#ifdef DEBUG
            if(lv->DataIndexTable[nexp-1].NodeList[lv->nodelistindex] != -1)
                endrun(1, "Current nodelist %ld entry (%d) not empty!\n", lv->nodelistindex, lv->DataIndexTable[nexp-1].NodeList[lv->nodelistindex]);
#endif
            lv->DataIndexTable[nexp-1].NodeList[lv->nodelistindex] = tw->tree->TopLeaves[no - tw->tree->lastnode].treenode;
            lv->nodelistindex++;
            return 0;
        }
    }
    /* out of buffer space. Need to interrupt. */
    if(lv->Nexport >= tw->BunchSize) {
        return -1;
    }
    lv->DataIndexTable[nexp].Task = task;
    lv->DataIndexTable[nexp].Index = target;
    lv->DataIndexTable[nexp].NodeList[0] = tw->tree->TopLeaves[no - tw->tree->lastnode].treenode;
    int i;
    for(i = 1; i < NODELISTLENGTH; i++)
        lv->DataIndexTable[nexp].NodeList[i] = -1;
    lv->Nexport++;
    lv->nodelistindex = 1;
    lv->NThisParticleExport++;
    return 0;
}

__device__ int force_treeev_shortrange_device(TreeWalkQueryGravShort * input,
        TreeWalkResultGravShort * output,
        LocalTreeWalk * lv, struct gravshort_tree_params * TreeParams_ptr, struct particle_data * particles)
{
    const ForceTree * tree = lv->tw->tree;
    const double BoxSize = tree->BoxSize;

    /*Tree-opening constants*/
    const double cellsize = GRAV_GET_PRIV(lv->tw)->cellsize;
    const double rcut = GRAV_GET_PRIV(lv->tw)->Rcut;
    const double rcut2 = rcut * rcut;
    const double aold = TreeParams_ptr->ErrTolForceAcc * input->OldAcc;
    const int TreeUseBH = TreeParams_ptr->TreeUseBH;
    double BHOpeningAngle2 = TreeParams_ptr->BHOpeningAngle * TreeParams_ptr->BHOpeningAngle;
    /* Enforce a maximum opening angle even for relative acceleration criterion, to avoid
     * pathological cases. Default value is 0.9, from Volker Springel.*/
    if(TreeUseBH == 0)
        BHOpeningAngle2 = TreeParams_ptr->MaxBHOpeningAngle * TreeParams_ptr->MaxBHOpeningAngle;

    /*Input particle data*/
    const double * inpos = input->base.Pos;

    /*Start the tree walk*/
    int listindex, ninteractions=0;

    /* Primary treewalk only ever has one nodelist entry*/
    for(listindex = 0; listindex < NODELISTLENGTH; listindex++)
    {
        int numcand = 0;
        /* Use the next node in the node list if we are doing a secondary walk.
         * For a primary walk the node list only ever contains one node. */
        int no = input->base.NodeList[listindex];
        int startno = no;
        if(no < 0)
            break;

        while(no >= 0)
        {
            /* The tree always walks internal nodes*/
            struct NODE *nop = &tree->Nodes[no];

            if(lv->mode == TREEWALK_GHOSTS && nop->f.TopLevel && no != startno)  /* we reached a top-level node again, which means that we are done with the branch */
                break;

            int i;
            double dx[3];
            for(i = 0; i < 3; i++)
                dx[i] = NEAREST(nop->mom.cofm[i] - inpos[i], BoxSize);
            const double r2 = dx[0] * dx[0] + dx[1] * dx[1] + dx[2] * dx[2];

            /* Discard this node, move to sibling*/
            if(shall_we_discard_node_device(nop->len, r2, nop->center, inpos, BoxSize, rcut, rcut2))
            {
                no = nop->sibling;
                /* Don't add this node*/
                continue;
            }

            /* This node accelerates the particle directly, and is not opened.*/
            int open_node = shall_we_open_node_device(nop->len, nop->mom.mass, r2, nop->center, inpos, BoxSize, aold, TreeUseBH, BHOpeningAngle2);

            if(!open_node)
            {
                /* ok, node can be used */
                no = nop->sibling;
                if(lv->mode != TREEWALK_TOPTREE) {
                    /* Compute the acceleration and apply it to the output structure*/
                    apply_accn_to_output_device(output, dx, r2, nop->mom.mass, cellsize);
                }
                continue;
            }

            if(lv->mode == TREEWALK_TOPTREE) {
                if(nop->f.ChildType == PSEUDO_NODE_TYPE) {
                    /* Export the pseudo particle*/
                    if(-1 == treewalk_export_particle_device(lv, nop->s.suns[0]))
                        return -1;
                    /* Move sideways*/
                    no = nop->sibling;
                    continue;
                }
                /* Only walk toptree nodes here*/
                if(nop->f.TopLevel && !nop->f.InternalTopLevel) {
                    no = nop->sibling;
                    continue;
                }
                no = nop->s.suns[0];
            }
            else {
                /* Now we have a cell that needs to be opened.
                * If it contains particles we can add them directly here */
                if(nop->f.ChildType == PARTICLE_NODE_TYPE)
                {
                    /* Loop over child particles*/
                    for(i = 0; i < nop->s.noccupied; i++) {
                        int pp = nop->s.suns[i];
                        lv->ngblist[numcand++] = pp;
                    }
                    no = nop->sibling;
                }
                else if (nop->f.ChildType == PSEUDO_NODE_TYPE)
                {
                    /* Move to the sibling (likely also a pseudo node)*/
                    no = nop->sibling;
                }
                else //NODE_NODE_TYPE
                    /* This node contains other nodes and we need to open it.*/
                    no = nop->s.suns[0];
            }
        }
        int i;
        for(i = 0; i < numcand; i++)
        {
            int pp = lv->ngblist[i];
            double dx[3];
            int j;
            for(j = 0; j < 3; j++)
                dx[j] = NEAREST(particles[pp].Pos[j] - inpos[j], BoxSize);
            const double r2 = dx[0] * dx[0] + dx[1] * dx[1] + dx[2] * dx[2];
            /* Compute the acceleration and apply it to the output structure*/
            apply_accn_to_output_device(output, dx, r2, particles[pp].Mass, cellsize);
        }
        ninteractions = numcand;
    }
    treewalk_add_counters_device(lv, ninteractions);
    return 1;
}

__device__ static MyFloat
grav_get_abs_accel_device(struct particle_data * PP, const double G)
{
    double aold=0;
    int j;
    for(j = 0; j < 3; j++) {
       double ax = PP->FullTreeGravAccel[j] + PP->GravPM[j];
       aold += ax*ax;
    }
    return sqrt(aold) / G;
}

__device__ static void
grav_short_copy_device(int place, TreeWalkQueryGravShort * input, TreeWalk * tw, struct particle_data *particles)
{
    input->OldAcc = grav_get_abs_accel_device(&particles[place], GRAV_GET_PRIV(tw)->G);
}

__device__ static void
treewalk_init_query_device(TreeWalk *tw, TreeWalkQueryBase *query, int i, const int *NodeList, struct particle_data *particles) {
    // Access particle data through particles argument
    for(int d = 0; d < 3; d++) {
        query->Pos[d] = particles[i].Pos[d];  // Use particles instead of P macro
    }

    if (NodeList) {
        memcpy(query->NodeList, NodeList, sizeof(query->NodeList[0]) * NODELISTLENGTH);
    } else {
        query->NodeList[0] = tw->tree->firstnode;  // root node
        query->NodeList[1] = -1;  // terminate immediately
    }
    TreeWalkQueryGravShort * query_short;
    // point query_short to the query
    query_short = (TreeWalkQueryGravShort *) query;
    // tw->fill(i, query, tw);
    grav_short_copy_device(i, query_short, tw, particles);
}

__device__ static void
treewalk_init_result_device(TreeWalk *tw, TreeWalkResultBase *result, TreeWalkQueryBase *query) {
    memset(result, 0, tw->result_type_elsize);  // Initialize the result structure
}

__device__ static void
grav_short_reduce_device(int place, TreeWalkResultGravShort * result, enum TreeWalkReduceMode mode, TreeWalk * tw, struct particle_data *particles)
{
    TREEWALK_REDUCE(GRAV_GET_PRIV(tw)->Accel[place][0], result->Acc[0]);
    TREEWALK_REDUCE(GRAV_GET_PRIV(tw)->Accel[place][1], result->Acc[1]);
    TREEWALK_REDUCE(GRAV_GET_PRIV(tw)->Accel[place][2], result->Acc[2]);
    if(tw->tree->full_particle_tree_flag)
        TREEWALK_REDUCE(particles[place].Potential, result->Potential);
}

__device__ void
treewalk_reduce_result_device(TreeWalk *tw, TreeWalkResultBase *result, int i, enum TreeWalkReduceMode mode, struct particle_data *particles) {
    // if (tw->reduce != NULL) {
    //     tw->reduce(i, result, mode, tw);  // Call the reduce function
    // }
    grav_short_reduce_device(i, (TreeWalkResultGravShort *) result, mode, tw, particles);
}

__global__ void treewalk_kernel(TreeWalk *tw, struct particle_data *particles, int *workset, size_t workset_size, struct gravshort_tree_params * TreeParams_ptr, unsigned long long int *maxNinteractions, unsigned long long int *minNinteractions, unsigned long long int *Ninteractions, double GravitySoftening) {
    GravitySoftening_device = GravitySoftening;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("FractionalGravitySoftening (__global__): %f\n", TreeParams_ptr->FractionalGravitySoftening);
    }

    if (tid < workset_size) {
        int i = workset[tid];

        TreeWalkQueryBase input;
        TreeWalkResultBase output;
        // Initialize query and result using device functions
        treewalk_init_query_device(tw, &input, i, NULL, particles);
        treewalk_init_result_device(tw, &output, &input);

        // Perform treewalk for particle
        LocalTreeWalk lv;
        lv.target = i;
        // tw->visit(&input, &output, &lv);
        force_treeev_shortrange_device((TreeWalkQueryGravShort*) &input, (TreeWalkResultGravShort*) &output, &lv, TreeParams_ptr, particles);
        // Reduce results for this particle
        treewalk_reduce_result_device(tw, &output, i, TREEWALK_PRIMARY, particles);

        // Update interactions count using atomic operations
        atomicAdd(Ninteractions, lv.Ninteractions);
        atomicMax(maxNinteractions, lv.maxNinteractions);
        atomicMin(minNinteractions, lv.minNinteractions);
    }
}

__global__ void test_kernel(TreeWalk *tw) {
    // printf("tw->tree->moments_computed_flag: %d\n", tw->tree->moments_computed_flag);
    printf("tw->WorkSet[0]: %d\n", tw->WorkSet[0]);
}

// Function to launch kernel (wrapper)
void run_treewalk_kernel(TreeWalk *tw, struct particle_data *particles, int *workset, size_t workset_size, struct gravshort_tree_params * TreeParams_ptr, double GravitySoftening, unsigned long long int *maxNinteractions, unsigned long long int *minNinteractions, unsigned long long int *Ninteractions) {
    
    int threadsPerBlock = 256;
    int blocks = (workset_size + threadsPerBlock - 1) / threadsPerBlock;
    // treewalk_kernel<<<blocks, threadsPerBlock>>>(tw, particles, workset, workset_size, TreeParams_ptr, maxNinteractions, minNinteractions, Ninteractions, GravitySoftening);
    // hipDeviceSynchronize();
    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     printf("CUDA error: %s\n", hipGetErrorString(err));
    // }
    printf("workset[0]: %d\n", workset[0]);
    printf("tw->WorkSet[0]: %d\n", tw->WorkSet[0]);
    test_kernel<<<1, 1>>>(tw);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    fflush(stdout);
}
