#include "hip/hip_runtime.h"
// pm_kernel.cu
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "box_iterator.hpp"
#include "petapm.h"


/* unnormalized sinc function sin(x) / x */
__device__ double sinc_unnormed(double x) {
    if(x < 1e-5 && x > -1e-5) {
        double x2 = x * x;
        return 1.0 - x2 / 6. + x2  * x2 / 120.;
    } else {
        return sin(x) / x;
    }
}


/* the transfer functions for force in fourier space applied to potential */
/* super lanzcos in CH6 P 122 Digital Filters by Richard W. Hamming */
__device__ double diff_kernel(double w) {
/* order N = 1 */
/*
 * This is the same as GADGET-2 but in fourier space:
 * see gadget-2 paper and Hamming's book.
 * c1 = 2 / 3, c2 = 1 / 12
 * */
    return 1 / 6.0 * (8 * sin (w) - sin (2 * w));
}


__global__
void potential_transfer_kernel(BoxIterator<hipfftComplex> begin, BoxIterator<hipfftComplex> end, PetaPM *pm) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    begin += tid;

    if (begin < end) {
        // Get global 3D coordinates of the current element in real space
        int x = begin.x();
        int y = begin.y();
        int z = begin.z();

        // Compute the corresponding wave numbers (kx, ky, kz), in grid unit
        int kx = x<=pm->Nmesh/2 ? x : (x-pm->Nmesh);
        int ky = y<=pm->Nmesh/2 ? y : (y-pm->Nmesh);
        int kz = z<=pm->Nmesh/2 ? z : (z-pm->Nmesh);
        int64_t k2 = 0.0;
        k2 += ((int64_t)kx) * kx;
        k2 += ((int64_t)ky) * ky;
        k2 += ((int64_t)kz) * kz;
        
        const double asmth2 = pow((2 * M_PI) * pm->Asmth / pm->Nmesh, 2);
        double f = 1.0;
        const double smth = exp(-k2 * asmth2) / k2;
        const double pot_factor = -pm->G / (M_PI * pm->BoxSize);

        int kpos[3] = {kx, ky, kz};
        // Apply CIC deconvolution
        for (int k = 0; k < 3; k++) {
            double tmp = (kpos[k] * M_PI) / pm->Nmesh;
            tmp = sinc_unnormed(tmp);
            f *= 1.0 / (tmp * tmp);
        }
        const double fac = pot_factor * smth * f * f;
        //CUDA TODO: add massive neutrino back

        // Handle zero mode separately
        if (k2 == 0) {
            begin->x = 0.0;
            begin->y = 0.0;
            return;
        }
        // Apply scaling factor
        begin->x *= fac;
        begin->y *= fac;
    }
}


extern "C" void launch_potential_transfer(Box3D box_complex, hipfftComplex* data, int rank, int size, PetaPM *pm, hipStream_t stream) {
    auto [begin_d, end_d] = BoxIterators(box_complex, data);
    const size_t num_elements = std::distance(begin_d, end_d);
    const size_t num_threads  = 256;
    const size_t num_blocks   = (num_elements + num_threads - 1) / num_threads;
    potential_transfer_kernel<<<num_blocks, num_threads, 0, stream>>>(begin_d, end_d, pm);
}




// static void force_transfer(PetaPM * pm, int k, hipfftComplex * value) {
//     double tmp0;
//     double tmp1;
//     /*
//      * negative sign is from force_x = - Del_x pot
//      *
//      * filter is   i K(w)
//      * */
//     double fac = -1 * diff_kernel (k * (2 * M_PI / pm->Nmesh)) * (pm->Nmesh / pm->BoxSize);
//     tmp0 = - value[0].y * fac;
//     tmp1 = value[0].x * fac;
//     value[0].x = tmp0;
//     value[0].y = tmp1;
// }
// static void force_x_transfer(PetaPM * pm, int64_t k2, int kpos[3], hipfftComplex * value) {
//     force_transfer(pm, kpos[0], value);
// }
// static void force_y_transfer(PetaPM * pm, int64_t k2, int kpos[3], hipfftComplex * value) {
//     force_transfer(pm, kpos[1], value);
// }
// static void force_z_transfer(PetaPM * pm, int64_t k2, int kpos[3], hipfftComplex * value) {
//     force_transfer(pm, kpos[2], value);
// }
// static void readout_potential(PetaPM * pm, int i, double * mesh, double weight) {
//     P[i].Potential += weight * mesh[0];
// }
// static void readout_force_x(PetaPM * pm, int i, double * mesh, double weight) {
//     P[i].GravPM[0] += weight * mesh[0];
// }
// static void readout_force_y(PetaPM * pm, int i, double * mesh, double weight) {
//     P[i].GravPM[1] += weight * mesh[0];
// }
// static void readout_force_z(PetaPM * pm, int i, double * mesh, double weight) {
//     P[i].GravPM[2] += weight * mesh[0];
// }






